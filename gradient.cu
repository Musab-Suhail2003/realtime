#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <math.h>

#define WIDTH 800
#define HEIGHT 600
#define GRAVITY 9.81

__device__ float dot(const float3& a, const float3& b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float3 normalize(const float3& v) {
    float len = sqrtf(dot(v, v));
    return make_float3(v.x / len, v.y / len, v.z / len);
}

__device__ bool hitSphere(float3 center, float radius, float3 rayOrigin, float3 rayDir, float* t) {
    float3 oc = make_float3(rayOrigin.x - center.x, rayOrigin.y - center.y, rayOrigin.z - center.z);
    float a = dot(rayDir, rayDir);
    float b = 2.0f * dot(oc, rayDir);
    float c = dot(oc, oc) - radius * radius;
    float discriminant = b * b - 4 * a * c;
    if (discriminant < 0) {
        return false;
    } else {
        *t = (-b - sqrtf(discriminant)) / (2.0f * a);
        return *t > 0.0f;
    }
}

__global__ void render(uchar4* pixels, float sphereY) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    float u = (2.0f * x) / (float)(WIDTH - 1) - 1.0f;
    float v = (2.0f * y) / (float)(HEIGHT - 1) - 1.0f;
    float aspect = WIDTH / (float)HEIGHT;

    float3 rayOrigin = make_float3(0.0f, 0.0f, 0.0f);
    float3 rayDir = make_float3(u * aspect, v, -1.0f);
    rayDir = normalize(rayDir);

    float t;
    float3 sphereCenter = make_float3(0.0f, sphereY, -1.0f);
    float radius = 0.1f;

    unsigned char r, g, b;

    if (hitSphere(sphereCenter, radius, rayOrigin, rayDir, &t)) {
        float3 hitPoint = make_float3(rayOrigin.x + t * rayDir.x,
                                     rayOrigin.y + t * rayDir.y,
                                     rayOrigin.z + t * rayDir.z);

        float3 normal = make_float3((hitPoint.x - sphereCenter.x) / radius,
                                    (hitPoint.y - sphereCenter.y) / radius,
                                    (hitPoint.z - sphereCenter.z) / radius);

        float3 lightDir = make_float3(1.0f, 1.0f, -1.0f);
        lightDir = normalize(lightDir);

        float diff = fmaxf(dot(normal, lightDir), 0.0f);
        diff = sqrtf(diff);

        r = (unsigned char)(diff * 255);
        g = (unsigned char)(diff * 0.5f * 255);
        b = (unsigned char)(diff * 0.7f * 255);
    } else {
        float t = 0.5f * (v + 1.0f);
        r = (unsigned char)((1.0f * (1.0f - t) + 0.5f * t) * 255);
        g = (unsigned char)((1.0f * (1.0f - t) + 0.7f * t) * 255);
        b = (unsigned char)((1.0f * (1.0f - t) + 1.0f * t) * 255);
    }

    int idx = y * WIDTH + x;
    pixels[idx] = make_uchar4(r, g, b, 255);
}

GLuint createTexture() {
    GLuint tex;
    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, WIDTH, HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    return tex;
}

int main(int argc, char** argv) {
    float time = 0.0f;
    float initialY = 1.0f; 
    float velocity = 0.0f;
    float damping = 0.95f; // Damping factor for energy loss on bounce
    float mass = 1.0f; // Mass of the sphere (in kg)
    float collisionTime = 0.01f; // Approximate collision duration (in seconds)

    if (argc > 1) {
        damping = atof(argv[1]); // Set initial position from command line argument
    } else {
        printf("No damping factor provided, using default value of 0.95\n");
    }

    if (!glfwInit()) {
        printf("Failed to init GLFW!\n");
        return -1;
    }

    GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "CUDA + OpenGL", NULL, NULL);
    glfwMakeContextCurrent(window);

    glewInit();

    GLuint tex = createTexture();

    hipGraphicsResource* cudaTexResource;
    hipGraphicsGLRegisterImage(&cudaTexResource, tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);

    dim3 blockSize(16, 16);
    dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, 
                  (HEIGHT + blockSize.y - 1) / blockSize.y);

    while (!glfwWindowShouldClose(window)) {
        time += 0.016f; // ~60fps

        // Update sphere's position using gravity
        float sphereY = initialY + velocity * time - 0.5f * GRAVITY * time * time;

        // Check if the sphere hits the bottom boundary
        if (sphereY < -0.865f) {
            sphereY = -0.865f; // Clamp position to the bottom boundary

            // Ensure collisionTime and mass are valid
            if (collisionTime <= 0.0f) collisionTime = 0.01f; // Prevent division by zero
            if (mass <= 0.0f) mass = 1.0f; // Prevent division by zero

            // Calculate the normal force
            float normalForce = (mass * fabs(velocity)) / collisionTime;
           
            // Calculate the reverse velocity based on damping
            float reverseVelocity = -velocity * damping;

            // Update the sphere's velocity
            velocity = reverseVelocity;

            // Reset initial position and time
            initialY = sphereY;
            time = 0.0f;
        }

        // Check if the sphere hits the top boundary
        if (sphereY > 1.0f) {
            sphereY = 1.0f; // Clamp position to the top boundary
            velocity = -velocity * damping; // Reverse and dampen velocity
            initialY = sphereY; // Reset initial position
            time = 0.0f; // Reset time
        }

        // Update velocity for the next frame
        velocity -= GRAVITY * 0.016f;

        hipArray_t array;
        hipGraphicsMapResources(1, &cudaTexResource, 0);
        hipGraphicsSubResourceGetMappedArray(&array, cudaTexResource, 0, 0);

        hipResourceDesc resDesc = {};
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = array;

        uchar4* devPtr;
        size_t size;
        hipMallocPitch((void**)&devPtr, &size, WIDTH * sizeof(uchar4), HEIGHT);

        render<<<gridSize, blockSize>>>(devPtr, sphereY);

        hipMemcpy2DToArray(array, 0, 0, devPtr, WIDTH * sizeof(uchar4), WIDTH * sizeof(uchar4), HEIGHT, hipMemcpyDeviceToDevice);

        hipFree(devPtr);
        hipGraphicsUnmapResources(1, &cudaTexResource, 0);

        glClear(GL_COLOR_BUFFER_BIT);
        glEnable(GL_TEXTURE_2D);
        glBindTexture(GL_TEXTURE_2D, tex);

        glBegin(GL_QUADS);
        glTexCoord2f(0.0f, 0.0f); glVertex2f(-1.0f, -1.0f);
        glTexCoord2f(1.0f, 0.0f); glVertex2f(1.0f, -1.0f);
        glTexCoord2f(1.0f, 1.0f); glVertex2f(1.0f, 1.0f);
        glTexCoord2f(0.0f, 1.0f); glVertex2f(-1.0f, 1.0f);
        glEnd();

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    hipGraphicsUnregisterResource(cudaTexResource);
    glfwDestroyWindow(window);
    glfwTerminate();

    return 0;
}

